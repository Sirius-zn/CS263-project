#include "hip/hip_runtime.h"
#include "cuda_utils.h"

#define MAX_EBD 5000
#define MAX_NB 10000
#define MAX_WK 1
#define Part_Size 1

#define thread_per_block 1024 // 128
#define point_per_block thread_per_block // 128

// pull-based aggregration with part-level sync and partial aggregration
__global__ void gcn_sag_forward_cuda(
    unsigned int numParts,
    int *nodePointer, // needs to be int * instead of unsigned int. undefined symbol bug otherwise.
    unsigned int ebd_dim,
    unsigned int numNodes,
    int *partNodePointer,
    int *edgeList,
    float *embed1,
    float *embed2
){
    int partId = blockDim.x * blockIdx.x + threadIdx.x;

    // printf("partID: %d \n", partId);
    // printf("numParts: %d \n", numParts);

    if(partId < numParts)
    {
        // printf("partID: %d \n", partId);
        // printf("numParts: %d \n", numParts);

        float currPt[MAX_EBD];
        int id = partNodePointer[partId*2+0]; // node idx
        int part = partNodePointer[partId*2+1]; // node part

        // Opt1: caching current node
        // #ifdef UNROLL
        // 	#pragma unroll
        // #endif
        for (int d = 0; d < ebd_dim; d++)
            if (part == 0)
                currPt[d] = embed2[id * ebd_dim + d];
            else
                currPt[d] = 0;

        int thisPointer = nodePointer[id];
        int degree = edgeList[thisPointer];
        int nid;
        int thisNumParts;

        if(degree % Part_Size == 0)
            thisNumParts = degree / Part_Size ;
        else
            thisNumParts = degree / Part_Size + 1;

        // #ifdef CONTIOUS
        // int base = thisPointer + part * Part_Size + 1;
        // #else
        int base = thisPointer + part + 1;
        // #endif

        // Opt3: caching neighbors idxs
        unsigned int nb_ebd_idx[MAX_NB];

        // #ifdef UNROLL
        //  #pragma unroll
        // #endif
        for(int i = 0; i < Part_Size; i++)
        {
            // #ifdef CONTIOUS
            //  if(i + part * Part_Size >= degree) break;
            //  nid = base + i;
            // #else
            // printf("%d\n", part + i * numParts);
            if(part + i * thisNumParts >= degree) break;
            nid = base + i * thisNumParts;
            // #endif

            nb_ebd_idx[i] = edgeList[nid] * ebd_dim;
            // printf("partID: %d , %d, %d, %d\n", partId, id, nid, edgeList[nid]);
            for (int d = 0; d < ebd_dim; d++)
                currPt[d] += embed2[nb_ebd_idx[i] + d];
        }

        // for (int d = 0; d < ebd_dim; d++)
        //  printf("%.3f ", currPt[d]);
        for (int d = 0; d < ebd_dim; d++)
            atomicAdd(&embed1[id * ebd_dim + d], currPt[d]);
    }
}

void gcn_sag_kernel_wrapper(
    unsigned int numParts,
    int *nodePointer,
    unsigned int ebd_dim,
    unsigned int numNodes,
    int *partNodePointer,
    int *edgeList,
    unsigned int num_of_edges,
    float *embed1, // output
    float *embed2 // input
) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    gcn_sag_forward_cuda<<<num_of_edges, 1024, 0, stream>>>(numParts, nodePointer, ebd_dim, numNodes, partNodePointer, edgeList, embed1, embed2);
    CUDA_CHECK_ERRORS();
}