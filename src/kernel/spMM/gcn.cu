#include "hip/hip_runtime.h"
#include "common.h"
#include "graph_reader/graph.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            float *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0f;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0f)
            {
                totalNnz++;
            }
        }
    }
    
    // std::cout << "sparsity: " << (totalNnz * 1.0) / (M * N) << std::endl;
    *outA = A;
    return totalNnz;
}


__global__ void Relu(float *matrix, int noOfElements){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < noOfElements)
        if (matrix[i] > 0)
            matrix[i] = matrix[i];
        else
            matrix[i] = 0;
}


int main(int argc, char **argv)
{
    float *X;
    float *dC, *dC1;
    float *W_1, *W_2;
    float *dh, *dh1;

    float *dCsrValA;
    int *dCsrRowPtrA;
    int *dCsrColIndA;
    int totalANnz;

    float *CsrValA;
    int *CsrRowPtrA;
    int *CsrColIndA;
    
    float alpha = 1.0f;
    float beta_1 = 1.0f;
    float beta_2 = 1.0f;
    
    hipsparseHandle_t handle = 0;
    hipblasHandle_t handle_blas = 0;
    hipsparseMatDescr_t Adescr = 0;
    
    // std::cout<<"Input: ./exe beg csr weight indim hiddendim outdim\n";
	if(argc != 8){
        std::cout<<"Wrong input\n"; 
        return -1;
    }
	
	const char *beg_file=argv[1];
	const char *csr_file=argv[2];
    const char *weight_file=argv[3];
    
    int N;
    int D = atoi(argv[4]);
    int K = atoi(argv[5]);
    int F = atoi(argv[6]);

    graph<long, long, int, int, int, float>
	*ginst = new graph<long, long, int, int, int, float>(beg_file, csr_file, weight_file);
    
    CsrRowPtrA = ginst->beg_pos;
    CsrColIndA = ginst->csr;
    CsrValA = ginst->weight;

    totalANnz = ginst -> edge_count;
    printf("nonZeros: %d\n", totalANnz);

    N = ginst-> vert_count;

    // Generate input
    // srand(9384);
    // generate_random_dense_matrix(N, D, &B);
    // B = (float *)malloc(sizeof(float) * N * D);
    // C = (float *)malloc(sizeof(float) * N * D);
    // memset(B, 0x01, sizeof(float) * N * D);

    // Create the cuSPARSE handle
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    // Create the cuBLAS handle
    CHECK_CUBLAS(hipblasCreate(&handle_blas));

    // Allocate device memory for vectors and the dense form of the matrix A
    CHECK(hipMalloc((void **)&X, sizeof(float) * N * D));

    // aggregration step param
    CHECK(hipMalloc((void **)&dC, sizeof(float) * N * K));
    CHECK(hipMalloc((void **)&dC1, sizeof(float) * N * F));

    // hidden parameters == scaled bias matrix (1 x K --> N x K)
    CHECK(hipMalloc((void **)&dh, sizeof(float) * N * K));
    CHECK(hipMalloc((void **)&dh1, sizeof(float) * N * F));

    // combination step param
    CHECK(hipMalloc((void **)&W_1, sizeof(float) * D * K));
    CHECK(hipMalloc((void **)&W_2, sizeof(float) * K * F));

    // Allocate device memory to store the sparse CSR representation of A
    CHECK(hipMalloc((void **)&dCsrValA, sizeof(float) * totalANnz));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (N + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz));

    CHECK(hipMemcpy(dCsrValA, CsrValA, sizeof(float) * totalANnz, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dCsrRowPtrA, CsrRowPtrA, sizeof(int) * (N + 1), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dCsrColIndA, CsrColIndA, sizeof(int) * totalANnz, hipMemcpyHostToDevice));

    // Construct a descriptor of the matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&Adescr));
    CHECK_CUSPARSE(hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO)); // HIPSPARSE_INDEX_BASE_ZERO

    float total_agg = 0;
    float total_update = 0;
    float total_update1 = 0;

    float milliseconds = 0;
    int iteration = atoi(argv[7]);

    for (int i = 0; i < iteration; i++)
    {
        // -------------------------------------------
        // Layer - 1 ---------------------------------
        // -------------------------------------------
        printf("iteration: %d\n", i);
        hipEvent_t start, stop;

        //////////////////////////////////////////////////
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));

        // Layer1 = Update step dgemm (N x D dot D x K --> N x K)
        hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, N, K, D, &alpha, X, N, W_1, D, &beta_2, dC, N);
        
        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        
        milliseconds = 0;
        CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        total_update += milliseconds;

        //////////////////////////////////////////////////
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        
        // Aggregration step spMM (dh = A.dC == N x N dot N x K --> N x K)
        // formula: alpha * A.dot(dC) + beta * dh  
        hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, K, N, totalANnz, &alpha, Adescr, dCsrValA, dCsrRowPtrA, dCsrColIndA, dC, N, &beta_1, dh, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        
        milliseconds = 0;
        CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        total_agg += milliseconds;

        //////////////////////////////////////////////////
        // dh = ReLU(dh)
        int noTrd = 1024;
        int noBlocks=(N * K)/1024 + 1;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));

        Relu<<<noBlocks, noTrd >>>(dh, N * K);
        
        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        
        milliseconds = 0;
        CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        total_update1 += milliseconds;

        // -------------------------------------------
        // Layer - 2 ---------------------------------
        // -------------------------------------------
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));

        // Layer2 = Update step dgemm  
        // ((dh) N x K dot (W2) K x F --> (dC1) N x F)
        hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, N, F, K, &alpha, dh, N, W_2, K, &beta_2, dC1, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        
        milliseconds = 0;
        CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        total_update += milliseconds;

        /////////////////////////////////////////////////
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        
        // Aggregration step spMM (A (N x N) dot dC (N x F) --> dh1 (N x F))
        // Formula: alpha * A.dot(dC1) + beta1 * dh1  
        hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, F, N, totalANnz, &alpha, Adescr, dCsrValA, dCsrRowPtrA, dCsrColIndA, dC1, N, &beta_1, dh1, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));

        milliseconds = 0;
        CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        total_agg += milliseconds;
    }

    // printf("Sparsity: %f %%\n", totalANnz * 1.0 / ((long)(N) * (long)(N)) * 100);
    float total = total_agg + total_update + total_update1;
    printf("Dim Reduction: %.3f %%\n", total_update/total * 100);
    printf("Aggre: %.3f %%\n", total_agg/total * 100);
    printf("Node Update: %.3f %%\n", total_update1/total * 100);
    printf("Time: %.3f ms\n", total);
    
    float total_ops = 2 * ((float)N * (float)D * (float)K + (float)N * (float)K * (float)F) + (float)totalANnz * (K + F);
    printf("Throughput: %.3f GFLOPS\n",  total_ops/(milliseconds / 1000.)/1e9);
    
    printf("\n\n\n\n");

    CHECK(hipFree(X));
    CHECK(hipFree(dC));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Adescr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUBLAS(hipblasDestroy(handle_blas));
    return 0;
}
