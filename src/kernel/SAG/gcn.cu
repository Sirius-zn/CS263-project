#include "hipblas.h"
#include "./pr.hpp"

// #define RUN_TOY

#ifdef RUN_TOY
	#define USE_RAND
	#define PROF_DETAIL
#endif
// #define DEBUG

int main(int argc, char** argv)
{
	Timer t, t1;
	float duration;

	ArgumentParser arguments(argc, argv, false, true, true);
	if (!arguments.hasInput || !arguments.hasEmbeding){
		std::cout << "Usage! ./exe graph_adj graph_embedding" << std::endl;
	}
	Graph graph(arguments.input, arguments.embedFilePath, false);
	
#ifdef PROF_DETAIL
	t.Start();
	graph.ReadGraph();
	duration = t.Finish();
	std::cout << "=> Read Graph (edge) takes: " << duration << "(ms)" << std::endl;
	t.Start();

	#ifdef load_ebd_from_file
		graph.ReadEmbedding();
	#else
	
	#endif
	duration = t.Finish();
	std::cout << "=> Read Graph (embeddinig) takes: " << duration << "(ms)" << std::endl;
#else
	graph.ReadGraph();
	#ifdef load_ebd_from_file
		graph.ReadEmbedding();
	#else
#endif 
#endif

	std::cout << "---------------" << std::endl;
#ifdef DEBUG
	//validate the embedding reading
	// for(int i = 0; i < 10; i++){
	// 	Embedding ebd = graph.pt_embeds[i];
	// 	std::cout << "pid[" << i  << "]= (";
	// 	// std::cout << "dim: " << ebd.embed.size() << std::endl;
	// 	for(int d  = 0; d < ebd.embed.size(); d++)
	// 		std::cout << ebd.embed[d] << " ";
	// 	std::cout << ")" <<std::endl;
	// }
#endif

#ifdef PROF_DETAIL
	t.Start();
	VirtualGraph vGraph(graph);
	vGraph.MakeUGraph();
	duration = t.Finish();
	std::cout << "=> Make Virtual Graph takes: " << duration << "(ms)" << std::endl;
#else
	VirtualGraph vGraph(graph);
	vGraph.MakeUGraph();
	// return 0;
#endif
	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

#ifdef load_ebd_from_file
	uint ebd_dim = graph.pt_embeds[0].embed.size();  // get the size of node embedding
#else
	uint ebd_dim = arguments.ebdSize;
#endif
	// printf("ebd_dim, %d\n", ebd_dim);

	uint hidden_dim = arguments.hidden;
	uint output_dim = arguments.no_Class;

	if(arguments.hasDeviceID)
		hipSetDevice(arguments.deviceID);	

	hipFree(0);

	float *embed1, *embed2;
	embed1  = new float[num_nodes * ebd_dim];
	embed2  = new float[num_nodes * ebd_dim];
	
	float *weight1, *weight2;
	weight1 = new float[ebd_dim * hidden_dim];
	weight2 = new float[hidden_dim * output_dim];

	// float *hidden, *output;
	// hidden = new float[num_nodes * hidden_dim];
	// output = new float[num_nodes * output_dim];


#ifdef load_ebd_from_file
#pragma omp parallel for
	for(int i = 0; i < num_nodes; i++)
	{
		Embedding curr_pt = graph.pt_embeds[i];
		for(int d  = 0; d < ebd_dim; d++){
			embed1[i * ebd_dim + d] = 0;
			embed2[i * ebd_dim + d] = curr_pt.embed[d];
		}
	}
#else

#ifdef USE_RAND
#pragma omp parallel for
	for(int i = 0; i < num_nodes; i++)
	{
		for(int d  = 0; d < ebd_dim; d++){
			embed1[i * ebd_dim + d] = 0.0f;
			embed2[i * ebd_dim + d] = 1.0f; //static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/dataRange));
		}
	}

#pragma omp parallel for
	for(int i = 0; i < ebd_dim; i++)
		for(int d  = 0; d < hidden_dim; d++)
			weight1[i * hidden_dim + d] = 1.0f; // static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/dataRange));

#pragma omp parallel for
	for(int i = 0; i < hidden_dim; i++)
		for(int d  = 0; d < output_dim; d++)
			weight2[i * output_dim + d] = 1.0f; //static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/dataRange));
#endif

#endif

	unsigned int *d_nodePointer;
	unsigned int *d_edgeList;
	PartPointer *d_partNodePointer; 
	float *d_embed1;
	float *d_embed1_test;

	float *d_embed2;
	float *d_hidden;
	float *d_embed3;
	float *d_output;
	float *d_weight1, *d_weight2;

	// layer-1 param
	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_embed1, num_nodes * ebd_dim * sizeof(float)));
	gpuErrorcheck(hipMemcpy(d_embed1, embed2, num_nodes * ebd_dim * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMalloc(&d_embed1_test, num_nodes * ebd_dim * sizeof(float)));

	gpuErrorcheck(hipMalloc(&d_hidden, num_nodes * hidden_dim * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_embed2, num_nodes * hidden_dim * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_weight1, ebd_dim * hidden_dim * sizeof(float)));
	gpuErrorcheck(hipMemcpy(d_weight1, weight1, ebd_dim * hidden_dim * sizeof(float), hipMemcpyHostToDevice));

	// layer-2 param
	gpuErrorcheck(hipMalloc(&d_embed3, num_nodes * output_dim * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_output, num_nodes * output_dim * sizeof(float)));
	gpuErrorcheck(hipMalloc(&d_weight2, hidden_dim * output_dim * sizeof(float)));

	gpuErrorcheck(hipMemcpy(d_weight2, weight2, hidden_dim * output_dim * sizeof(float), hipMemcpyHostToDevice));

	// graph and virtual graph
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));

	// for(int i = 0; i < vGraph.numParts; i++){
	// 	std::cout << "part: " << i << "local part: " << vGraph.partNodePointer[i].part \
	// 	<< " node: " << vGraph.partNodePointer[i].node << std::endl;
	// }

	divide_node_by_block(vGraph);

	t1.Start();
	hipblasHandle_t handle_blas = 0;
	float alpha = 1.0f;
	float beta_1 = 1.0f;

	// std::cout << "vGraph.numParts: " << vGraph.numParts << std::endl;
	// for (unsigned int i = 0; i < vGraph.numParts; i++){
	// 	std::cout << "GID: " << i << " TNID: " << vGraph.partNodePointer[i].node << " LID: " << vGraph.partNodePointer[i].part << std::endl;
	// }

	// for (unsigned int i = 0; i < vGraph.numParts; i++){
	// 	std::cout << "GID: " << i << " TNID: " << vGraph.partNodePointer[i].node \
	// 			  << " LSAddr: " << vGraph.partNodePointer[i].node_shared_addr << std::endl;
		
	// 	if ((i + 1)%thread_per_block == 0)
	// 		std::cout << "============================" << std::endl;

	// layer-1
#ifdef PROF_DETAIL

#ifdef RUN_TOY
	kernel<<< (vGraph.numParts * MAX_WK)/thread_per_block + 1 , thread_per_block >>>(vGraph.numParts, 
																						d_nodePointer,
																						ebd_dim,
																						num_nodes,
																						d_partNodePointer,
																						d_edgeList,
																						d_embed1_test,
																						d_embed1
																						);

	gpuErrorcheck( hipDeviceSynchronize() );
	gpuErrorcheck(hipMemcpy(embed1, d_embed1_test, num_nodes * ebd_dim * sizeof(float), hipMemcpyDeviceToHost));
	for(int i = 0; i < num_nodes; i++)
	{
		for(int d  = 0; d < ebd_dim; d++){
			printf("%.3f ", embed1[i * ebd_dim + d]);
			// embed2[i * ebd_dim + d] = 1; //static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/dataRange));
		}
		std::cout << std::endl;
	}		
#else
	t.Start();
	hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, num_nodes, hidden_dim, ebd_dim, &alpha, d_embed1, num_nodes, d_weight1, ebd_dim, &beta_1, d_hidden, num_nodes);
	gpuErrorcheck( hipDeviceSynchronize() );
	duration = t.Finish();
	std::cout << "=> Layer-1 Dimension Reduction: " << duration << "(ms)" << std::endl;
	t.Start();
	kernel<<< (vGraph.numParts * MAX_WK)/thread_per_block + 1 , thread_per_block >>>(vGraph.numParts, 
																			d_nodePointer,
																			hidden_dim,
																			num_nodes,
																			d_partNodePointer,
																			d_edgeList,
																			d_embed2,
																			d_hidden
																			);
	duration = t.Finish();
	std::cout << "=> Layer-1 Aggregration: " << duration << "(ms)" << std::endl;
#endif

#else
	hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, num_nodes, hidden_dim, ebd_dim, &alpha, d_embed1, num_nodes, d_weight1, ebd_dim, &beta_1, d_hidden, num_nodes);

	kernel<<< (vGraph.numParts * MAX_WK)/thread_per_block + 1 , thread_per_block >>>(vGraph.numParts, 
																					d_nodePointer,
																					hidden_dim,
																					num_nodes,
																					d_partNodePointer,
																					d_edgeList,
																					d_embed2,
																					d_hidden
																					);
#endif

#ifndef RUN_TOY
	clearLabel<<< num_nodes * ebd_dim/thread_per_block + 1 , thread_per_block >>>( d_hidden,
																				   d_embed2,
																				   hidden_dim, 
																				   num_nodes
																					);
	Relu<<<num_nodes * hidden_dim/thread_per_block, thread_per_block >>>(d_hidden, num_nodes * hidden_dim);

	// layer-2
#ifdef PROF_DETAIL
	t.Start();
	hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, num_nodes, output_dim, hidden_dim, &alpha, d_hidden, num_nodes, d_weight2, hidden_dim, &beta_1, d_embed3, num_nodes);
	duration = t.Finish();
	gpuErrorcheck( hipDeviceSynchronize() );	
	std::cout << "=> Layer-2 Dimension Reduction: " << duration << "(ms)" << std::endl;
	t.Start();
	kernel<<< (vGraph.numParts * MAX_WK) /thread_per_block + 1 , thread_per_block >>>(vGraph.numParts, 
																			d_nodePointer, 
																			output_dim,
																			num_nodes,
																			d_partNodePointer,
																			d_edgeList,
																			d_output,
																			d_embed3);
	duration = t.Finish();
	gpuErrorcheck( hipDeviceSynchronize() );	
	std::cout << "=> Layer-2 Aggregration: " << duration << "(ms)" << std::endl;
#else
	hipblasSgemm(handle_blas, HIPBLAS_OP_N, HIPBLAS_OP_N, num_nodes, output_dim, hidden_dim, &alpha, d_hidden, num_nodes, d_weight2, hidden_dim, &beta_1, d_embed3, num_nodes);
	kernel<<< (vGraph.numParts * MAX_WK)/thread_per_block + 1 , thread_per_block >>>(vGraph.numParts, 
																			d_nodePointer, 
																			output_dim,
																			num_nodes,
																			d_partNodePointer,
																			d_edgeList,
																			d_output,
																			d_embed3);
#endif
	clearLabel<<< num_nodes * hidden_dim/thread_per_block + 1 , thread_per_block >>>(d_output,
																					d_embed3,
																					output_dim, 
																					num_nodes);
	gpuErrorcheck( hipPeekAtLastError() );
	gpuErrorcheck( hipDeviceSynchronize() );	
	hipDeviceSynchronize();

	float runtime = t1.Finish();
	cout << "**GCN " << runtime << " (ms).\n\n\n";
#endif

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_partNodePointer));
}
